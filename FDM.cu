
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
using namespace std;
// Fractional order model calculated on GPU with nvcc

#define ThreadPerBlock 1024
__global__
void add(int m, int n, double *Y, double *dx1, double *dx2, double *temp2)
{  
	dx1[m-1] = temp2[0];
	dx2[m-1] = temp2[1];
	__shared__ double cache[ThreadPerBlock]; 
	double temp = 0;
	int sindex = threadIdx.x;
	int stride = blockDim.x;
	for(int s = sindex; s <min(m,n); s+=stride)
	{
		if(s==0 || s==1)
		{
		
		}
		else
		{
			if(blockIdx.x == 0) temp += (Y[s] * dx1[m - s]);
			if(blockIdx.x == 1) temp += (Y[s] * dx2[m - s]);
		}
	}
	cache[sindex] = temp;
	__syncthreads();	
	int i = blockDim.x / 2; 
	while(i!=0)
	{
		if(sindex < i)
			cache[sindex] += cache[sindex + i];
		__syncthreads();
		i /= 2;
	}
	if(sindex == 0)
	{
		if(blockIdx.x == 0) temp2[0] = cache[0] * (-1); //dx1[m] = cache[0] * (-1);
		if(blockIdx.x == 1) temp2[1] = cache[0] * (-1); //dx2[m] = cache[0] * (-1);
	}
}
int main()
{
    //10-1024, 15-32768
    int N = 1 << 18; //19 //T
    int n = 1 << 15;      //L
    double a = 0.8;
    int ngpus = 4;
    const int NGPUS = 4;
    int const S = 8;
    int SS, P, s;
    double *d_Y[NGPUS], *d_dx[NGPUS * 2];
    double *h_Y, *h_dx[NGPUS * 2], *u, *y;
    double *A[S], *B, *C;
    double *d_temp[4], *h_temp[4];
    hipStream_t stream[NGPUS];
    // memory allocation
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipStreamCreate(&stream[i]);
        hipMalloc((void **)&d_Y[i], n * sizeof(double));
        hipMalloc((void **)&d_temp[i], 2 * sizeof(double));
        hipHostMalloc((void **)&h_temp[i], 2 * sizeof(double), hipHostMallocDefault);
        // two d_dx and h_dx for each calculation block on each GPU
        hipMalloc((void **)&d_dx[i * 2], N * sizeof(double));
        hipMalloc((void **)&d_dx[i * 2 + 1], N * sizeof(double));
        hipHostMalloc((void **)&h_dx[i * 2], N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc((void **)&h_dx[i * 2 + 1], N * sizeof(double), hipHostMallocDefault);
    }
    for (int st = 0; st < S; st++)
    {
        hipHostMalloc((void **)&A[st], S * sizeof(double), hipHostMallocDefault);
    }
    hipHostMalloc((void **)&B, S * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void **)&C, S * sizeof(double), hipHostMallocDefault);

    // set matrice data
    switch (S)
    {
    case 2: //2 order system parameters!!!!!!!!!!!!!!!!alfa
        A[0][0] = 0.7;
        A[0][1] = 0;
        A[1][0] = 1;
        A[1][1] = 0.4;
        B[0] = 1;
        B[1] = 0;
        C[0] = 0;
        C[1] = 1;
        break; //end of system/*/
    case 4:    //4 order system parameters
        A[0][0] = -2.1624 + 0.8, A[0][1] = -1.9225, A[0][2] = -0.7318, A[0][3] = -0.0859;
        A[1][0] = 1.0, A[1][1] = 0.8, A[1][2] = 0.0, A[1][3] = 0.0;
        A[2][0] = 0.0, A[2][1] = 1.0, A[2][2] = 0.8, A[2][3] = 0.0;
        A[3][0] = 0.0, A[3][1] = 0.0, A[3][2] = 1.0, A[3][3] = 0.8;
        B[0] = 1, B[1] = 0, B[2] = 0, B[3] = 0;
        C[0] = 0, C[1] = -1, C[2] = 0, C[3] = 0;
        break; //end of system/*/
    case 8:    //8order system parameters in progresss
        A[0][0] = -4.4557 + 0.8, A[0][1] = -8.5928, A[0][2] = -8.9663, A[0][3] = -5.2783, A[0][4] = -1.6870, A[0][5] = -0.2600, A[0][6] = -0.0171, A[0][7] = -0.0012;
        A[1][0] = 1.0, A[1][1] = 0.8, A[1][2] = 0.0, A[1][3] = 0.0, A[1][4] = 0.0, A[1][5] = 0.0, A[1][6] = 0.0, A[1][7] = 0.0;
        A[2][0] = 0.0, A[2][1] = 1.0, A[2][2] = 0.8, A[2][3] = 0.0, A[2][4] = 0.0, A[2][5] = 0.0, A[2][6] = 0.0, A[2][7] = 0.0;
        A[3][0] = 0.0, A[3][1] = 0.0, A[3][2] = 1.0, A[3][3] = 0.8, A[3][4] = 0.0, A[3][5] = 0.0, A[3][6] = 0.0, A[3][7] = 0.0;
        A[4][0] = 0.0, A[4][1] = 0.0, A[4][2] = 0.0, A[4][3] = 1.0, A[4][4] = 0.8, A[4][5] = 0.0, A[4][6] = 0.0, A[4][7] = 0.0;
        A[5][0] = 0.0, A[5][1] = 0.0, A[5][2] = 0.0, A[5][3] = 0.0, A[5][4] = 1.0, A[5][5] = 0.8, A[5][6] = 0.0, A[5][7] = 0.0;
        A[6][0] = 0.0, A[6][1] = 0.0, A[6][2] = 0.0, A[6][3] = 0.0, A[6][4] = 0.0, A[6][5] = 1.0, A[6][6] = 0.8, A[6][7] = 0.0;
        A[7][0] = 0.0, A[7][1] = 0.0, A[7][2] = 0.0, A[7][3] = 0.0, A[7][4] = 0.0, A[7][5] = 0.0, A[7][6] = 1.0, A[7][7] = 0.8;
        B[0] = 0, B[1] = 0, B[2] = 0, B[3] = 1, B[4] = 0, B[5] = 1, B[6] = 0, B[7] = 0;
        C[0] = 0, C[1] = -1, C[2] = 0, C[3] = 0, C[4] = 1, C[5] = 0, C[6] = 0, C[7] = 0;
        break; //end of system/*/
    }
    hipHostMalloc((void **)&h_Y, n * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void **)&u, N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void **)&y, N * sizeof(double), hipHostMallocDefault);

    // set u, and Y reset y
    for (int j = 0; j < N; j++)
    {
        u[j] = 1;
        y[j] = 0;
        h_dx[0][j] = 0;
        if (j < n)
        {
            if (j == 0)
            {
                h_Y[0] = 1;
            }
            else
            {
                if (j == 1)
                {
                    h_Y[1] = a;
                }
                else
                {
                    h_Y[j] = h_Y[j - 1] * ((a - j + 1) / j);
                    h_Y[j - 1] = h_Y[j - 1] * pow(-1, j + 1);
                }
            }
        }
    }
    h_Y[n - 1] = h_Y[n - 1] * (-1);
    u[0] = 0;

    //values for t=1
    for (SS = 0; SS < S; SS++)
    {
        h_dx[SS][1] = B[SS] * u[1];
        y[1] += C[SS] * h_dx[SS][1];
    }

    // data file
    ofstream myfile;
    myfile.open("FIND.txt", ios::app);

    // load Y to devices
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        if (i == 0)
        {
            hipMemcpyAsync(d_Y[0], h_Y, n * sizeof(double), hipMemcpyHostToDevice, stream[0]);
        }
        else
        {
            hipMemcpyAsync(d_Y[i], d_Y[0], n * sizeof(double), hipMemcpyDeviceToDevice, stream[i]);
        }
    }
    hipDeviceSynchronize(); //synchornize davices
    for (int i = 0; i < ngpus; i++)
    {
        h_temp[i][0] = h_dx[i * 2][1];
        h_temp[i][1] = h_dx[i * 2 + 1][1];
    }
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        hipMemcpyAsync(d_dx[i * 2], h_dx[i * 2], sizeof(double), hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_dx[i * 2 + 1], h_dx[i * 2 + 1], sizeof(double), hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_temp[i], h_temp[i], 2 * sizeof(double), hipMemcpyHostToDevice, stream[i]);
    }
    hipDeviceSynchronize();
    double t1, t2;
    // start calculating
    for (int m = 2; m < N; m++)
    {
        if (m >= N-10)
        { 
            t1 = omp_get_wtime();
        }
        for (int i = 0; i < ngpus; i++)
        {
            hipSetDevice(i);
            add<<<2, ThreadPerBlock, 0, stream[i]>>>(m, n, d_Y[i], d_dx[i * 2], d_dx[i * 2 + 1], d_temp[i]);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < ngpus; i++)
        {
            hipSetDevice(i);
            hipMemcpyAsync(h_temp[i], d_temp[i], 2 * sizeof(double), hipMemcpyDeviceToHost, stream[i]);
        }
        hipDeviceSynchronize();
        for (int i = 0; i < ngpus; i++)
        {
            h_dx[i * 2][m] = h_temp[i][0];
            h_dx[i * 2 + 1][m] = h_temp[i][1];
        }
        for (P = 0; P < S; P++)
        {
            for (s = 0; s < S; s++)
            {
                h_dx[P][m] += (A[P][s] * h_dx[s][m - 1]);
            }
            h_dx[P][m] += B[P] * u[m];
        }
        for (P = 0; P < S; P++)
        {
            y[m] += (C[P] * h_dx[P][m]);
        }
        for (int i = 0; i < ngpus; i++)
        {
            h_temp[i][0] = h_dx[i * 2][m];
            h_temp[i][1] = h_dx[i * 2 + 1][m];
        }
        for (int i = 0; i < ngpus; i++)
        {
            hipSetDevice(i);
            hipMemcpyAsync(d_temp[i], h_temp[i], 2 * sizeof(double), hipMemcpyHostToDevice, stream[i]);
        }
        hipDeviceSynchronize();
        if (m >= N-10)
        { 
            t2 = omp_get_wtime();
            cout << t2-t1 << endl; 
            myfile << t2-t1 << endl;
        }
    }

     //"debug" purpose
     for (int k = 0; k < 10; k++)
     {
         cout << y[k] << ' ' << k << endl;
     }
     cout << endl;
     for (int k = N / 2; k < N / 2 + 10; k++)
     {
         cout << y[k] << ' ' << k << endl;
     }
     cout << endl;
     for (int k = N - 10; k < N; k++)
     {
         cout << y[k] << ' ' << k << endl;
     }

    hipFree(h_Y);
    hipFree(d_Y);
    hipFree(h_dx);
    hipFree(d_dx);
    hipFree(u);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(y);

    // data save
     cout << "N=" << N << " n=" << n << " TpB=" << ThreadPerBlock << " NGPUS=" << NGPUS << endl;
    myfile.close();
    return 0;
}
