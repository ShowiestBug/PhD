
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
using namespace std;
//prototype single gpu divided dot product
//N-t
//n-L
#define ThreadPerBlock 256
__global__
void add(int m, int n, double *Y, double *u, double *p_dx)
{
	__shared__ double cache[ThreadPerBlock];
	double temp = 0;
	
  	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
  	int cacheIndex = threadIdx.x;
  	for (int i = index; i < min(m,n); i += stride)
  	{
  		temp += Y[i] * u[m-i];
	}
	cache[cacheIndex] = temp;
	__syncthreads();	
	int i = blockDim.x / 2; 
	while(i!=0)
	{
		if(cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if(cacheIndex == 0)
		p_dx[blockIdx.x] = cache[0];
}

__global__
void red(double *p_dx, int num)
{
	int idx = threadIdx.x;
	int i = num/ 2; 
	int ii = num % 2;
	while(i!=0)
	{
		if(ii!=0 && idx == 0)
		{
			p_dx[0] += p_dx[num-1];
		}
		if(idx < i)
			p_dx[idx] += p_dx[idx + i];
		__syncthreads();
		ii = i % 2;
		i /= 2;
	}
	
}
int main()
{
	double t1,t2,t3,t4;
	t1 = omp_get_wtime(); 
	
	int N = 1 << 19;//19
	int n = 1 << 17;//17
	double a = 0.5;
	
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	
	double *h_Y, *h_u, *ph_dx, *dx, *tep, *h_t;
	double *d_Y, *d_u, *pd_dx, *d_t;
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipMalloc((void **) &d_Y, n*sizeof(double));
	hipMalloc((void **) &d_u, N*sizeof(double));
	hipMalloc((void **) &pd_dx, numBlocks*sizeof(double));
	hipMalloc((void **) &d_t, 5*sizeof(double));
	
	
	hipHostMalloc((void **) &h_t, 5*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void **) &tep, 2*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void **) &ph_dx, numBlocks*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void **) &h_Y, n*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void **) &h_u, N*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void **) &dx, N*sizeof(double), hipHostMallocDefault);

	for(int j=0;j<N;j++)
	{
		h_u[j] = 1;
		dx[j] = 0;
		if(j<n)
		{
			if (j == 0)
			{
				h_Y[0] = 1;
			}
			else
			{
				if (j == 1)
				{
					h_Y[1] = a;
				}
				else
				{
					h_Y[j] = h_Y[j - 1] * ((a - j + 1) / j);
					h_Y[j - 1] = h_Y[j - 1] * pow(-1, j + 1);
				}
			}
		}
	}
	h_Y[n-1] = h_Y[n-1] * (-1);
	h_u[0] = 0;
    t3 = omp_get_wtime();
    hipMemcpyAsync(d_Y, h_Y, n*sizeof(double), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(d_u, h_u, N*sizeof(double), hipMemcpyHostToDevice, stream);
    for(int m=0;m<N;m++)
    {
    	numBlocks  = (m + blockSize) / blockSize;
   		add<<<numBlocks, blockSize>>>(m,n,d_Y,d_u,pd_dx);
		hipDeviceSynchronize();
		hipMemcpyAsync(ph_dx, pd_dx, numBlocks*sizeof(double), hipMemcpyDeviceToHost, stream);
		hipDeviceSynchronize();
		if(numBlocks == 1)
	   	{
	   		dx[m] = ph_dx[0];
	    }
	    else
	    {
			for(int k=0;k<numBlocks;k++)
			{
				dx[m]+=ph_dx[k];		
			}
		}
	}
    t4 = omp_get_wtime();
    for(int k=0;k<10;k++)
    {
		cout<<dx[k]<<' '<<k<<endl;//h_Y[k]<<' '<<h_u[k]<<endl;
	}
	cout<<endl;
	for(int k=N/2;k<N/2+10;k++)
    {
		cout<<dx[k]<<' '<<k<<endl;
	}
	cout<<endl;
	for(int k=N-10;k<N;k++)
    {
		cout<<dx[k]<<' '<<k<<endl;
	}
	cout<<endl;
    hipStreamDestroy(stream);
    hipFree(h_Y);
    hipFree(h_u);
    hipFree(dx);
    hipFree(d_Y);
    hipFree(d_u);
    hipFree(pd_dx);
    hipFree(ph_dx);
	t2 = omp_get_wtime();
	cout<<"N="<<N<<"\t"<<"n="<<n<<endl;
	cout<<t4-t3<<' '<<t2-t1<<endl;
	return 0;
}


