
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <ctime>
using namespace std;

int minimum(double *in, int n)
{	
	int k = 1;
 	int a = in[0];
	for(int i=1;i<n;i++)
	{
		if(a<in[i])
		{
			a = in[i];
			k = i;
		}
	}
	return k;
}
//__global__
//void add(int N, int n, double *Y, double *dx, double *u)
//{
//
//
//}
double fun(double *vec)
{
	double temp=0;
	for(int i=0;i<5;i++)
	{
		temp += vec[i];
	}
	return temp;
}
int main()
{
	int const nvars = 5;
	int numPop = 20;
	int numBlocks = 10;
	int IteCount = 0;
	double *Pop[nvars], *Score;
	double BestScore, *BestGenes;
	hipMallocManaged(&Score, numPop*numBlocks*sizeof(double));
	for(int n=0;n<nvars;n++)
	{
		hipMallocManaged(&Pop[n], numPop*numBlocks*sizeof(double));
	}
	int EliteKids = 3;
	int MutaKids = 6;
	int CrossParents = (2 * (numPop - EliteKids - MutaKids)) + MutaKids;
	int Pass = numBlocks - 1;
	if(Pass > 0.2 * numPop)
	{
        Pass = 0.2 * numPop;
	}
	srand((int)time(0));
	//random population
    for(int i=0;i<nvars;i++)
    {
    	for(int j=0;j<numPop*numBlocks;j++)
    	{
    		Pop[i][j] = (rand() % 1000 + 1);
    		Pop[i][j] /= 1000;
    	}
	}
    //score population
    double *vec;
    hipMallocManaged(&vec, nvars*sizeof(double));
    for(int i=0;i<numPop*numBlocks;i++)
    {
    	for(int j=0;j<nvars;j++)
    	{
    		vec[j] = Pop[j][i];
    	}
    	Score[i] = fun(vec);
	}
	//main loop
    int exitFlag = 0;
//    do
//    {
//    IteCount++;
	double a[] = {3,7,2,5,6,4,9,10};
	cout<<minimum(a,8);
//
//    exitFlag++;
//	}while(exitFlag==0);
	return 0;
}
